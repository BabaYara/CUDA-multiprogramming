
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

//Kernel
__global__  void DFA_kernal(int *t_m,int *in,int n_state,int n_sigma,int init_state,int final_state,int n,int *out) {
	extern __shared__ int state_vectors[];
	int i,j;
	int t_id = threadIdx.x;
	//int *state_vector = (int *)malloc(sizeof(int)*n_state);
	//TO-DO: give this thread a part of string
	for(i=0;i<n_state;i++){
		//state_vector[i] = t_m[n_sigma*i + in[t_id]];
		state_vectors[n_state*t_id + i] = t_m[n_sigma*i + in[t_id]];
	}
	__syncthreads();
	//O(P) reduction
	for(i = 1; i < blockDim.x; ++i) {
        __syncthreads();
        if(t_id == i) {
            for(j=0;j<n_state;j++){
            	state_vectors[n_state*t_id + j] = state_vectors[n_state*t_id + state_vectors[n_state*(t_id-1) + j]];
            }
        }
    }
    for(int i=0;i<n_state;i++){
    	out[i] = state_vectors[n_state*(n-1) + i];
    }

}
 
int main()
{
	//Variables
	int STATES,SIGMA,INITIAL_STATE,FINAL_STATE,INPUT_LENGTH;
	int i,j;
	//Taking input
	//cin >> STATES >> SIGMA >> FINAL_STATE >> INPUT_LENGTH;
	scanf("%d %d %d %d",&STATES,&SIGMA,&FINAL_STATE,&INPUT_LENGTH);
	//An additional state has to be added for complete transition function
	STATES++;
	INITIAL_STATE = 0;
	//Input memory allocation and input retrival
	int *input = (int *)malloc(sizeof(int)*INPUT_LENGTH);
	for(i=0;i<INPUT_LENGTH;i++){
		scanf("%d",&input[i]);
	}
	//Allocating memory and retriving to transition matrix
	int **transition_matrix = (int **)malloc(sizeof(int *)*STATES);
	int *transition_matrix_data = (int *)malloc(sizeof(int)*STATES*SIGMA);
	for(i=0;i<STATES;i++){
		transition_matrix[i] = &transition_matrix_data[i*SIGMA];
	}

	for(i=0;i<STATES;i++){
		for(j=0;j<SIGMA;j++){
			scanf("%d",&transition_matrix[i][j]);
		}
	}
	//printing the input taken
	for(i=0;i<INPUT_LENGTH;i++){
		printf("%d ",input[i]);
	}
	printf("\n");
	for(i=0;i<STATES;i++){
		for(j=0;j<SIGMA;j++){
			printf("%d ",transition_matrix[i][j]);
		}
		printf("\n");
	}
	printf("\n");
	int *h_out = (int *)malloc(sizeof(int)*STATES);
	//////////////////////////////////////////////////////////////////////////
 	//Device memory
	int *d_transition_matrix;
	int *d_input;
	int *d_output;
 	
 	//Allocating and initializing memory on GPU
	hipMalloc((void**)&d_transition_matrix,sizeof(int)*STATES*SIGMA);
	hipMemcpy((void *)d_transition_matrix,(void *)transition_matrix_data,sizeof(int)*STATES*SIGMA,hipMemcpyHostToDevice);
	
	hipMalloc((void**)&d_input,sizeof(int)*INPUT_LENGTH);
	hipMemcpy((void *)d_input,(void *)input,sizeof(int)*INPUT_LENGTH,hipMemcpyHostToDevice);

	hipMalloc((void**)&d_output,sizeof(int)*STATES);
	
	//Declaring grid and block size
	dim3 dimBlock(INPUT_LENGTH,1,1);
	dim3 dimGrid(1,1,1);
	DFA_kernal<<<dimGrid, dimBlock, STATES*INPUT_LENGTH>>>(d_transition_matrix,d_input,STATES,SIGMA,INITIAL_STATE,FINAL_STATE,INPUT_LENGTH,d_output);

	hipMemcpy((void *)h_out,(void *)d_output,sizeof(int)*STATES,hipMemcpyDeviceToHost);

	hipFree(d_output);
	hipFree(d_transition_matrix);
	hipFree(d_input);

	/*for(i=0;i<STATES;i++){
		printf("%d\n",h_out[i]);
	}*/
	if(h_out[0]==FINAL_STATE){
		printf("Automata is accepting the string\n");
	}
	else{
		printf("String not accepted\n");
	}

	printf("All done\n");
	return 0;
}